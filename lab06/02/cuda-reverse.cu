#include "hip/hip_runtime.h"
/****************************************************************************
 *
 * cuda-reverse.cu - Array reversal with CUDA
 *
 * Copyright (C) 2017--2022 by Moreno Marzolla <moreno.marzolla(at)unibo.it>
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *    http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 ****************************************************************************/

/***
% HPC - Array reversal with CUDA
% Moreno Marzolla <moreno.marzolla@unibo.it>
% Last updated: 2022-11-15

Write a program that reverses an array of length $n$, i.e., swaps the
content of position $0$ and $n-1$, then position $1$ and $n-2$ and so
on. Specifically, write two versions of such a program: the first
version reverses an input array `in[]` into a different output array
`out[]`, so that the input is not modified. The second version
reverses an array `in[]` "in place" using at most $O(1)$ additional
storage.

The file [cuda-reverse.cu](cuda-reverse.cu) provides a CPU-based
implementation of `reverse()` and `inplace_reverse()` functions.  You
are required to modify the functions to make use of the GPU.

**Hint:** `reverse()` can be easily transformed into a kernel executed
by $n$ CUDA threads (one for each element of the array). Each thread
copies an element from `in[]` to the correct position of `out[]`.  Use
one-dimensional _thread blocks_, since that makes easy to map threads
to array elements. The `inplace_reverse()` function can be transformed
into a kernel as well, but in this case only $\lfloor n/2 \rfloor$
CUDA threads are required (note the rounding): each thread swaps an
element from the first half of `in[]` with the appropriate element
from the second half. Make sure that the program works also when the
input length $n$ is odd.

To copmile:

        nvcc cuda-reverse.cu -o cuda-reverse

To execute:

        ./cuda-reverse [n]

Example:

        ./cuda-reverse

## Files

- [cuda-reverse.cu](cuda-reverse.cu)
- [hpc.h](hpc.h)

***/
#include "../hpc.h"
#include <stdio.h>
#include <math.h>
#include <assert.h>

#define BLKDIM  1024

__global__ void reverse_kernel(int *in, int *out, int n) {
    const int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
       out[index] = in[n - 1 - index];
    }
}

/* Reverses `in[]` into `out[]`. */
void reverse( int *in, int *out, int n )
{
    int *d_in, *d_out;

    const size_t size = n*sizeof(*in);

    cudaSafeCall( hipMalloc( (void **)&d_in, size ) );
    cudaSafeCall( hipMalloc( (void **)&d_out, size ) );

    cudaSafeCall(hipMemcpy(d_in, in, size, hipMemcpyHostToDevice));

    reverse_kernel<<<(n + BLKDIM - 1)/BLKDIM, BLKDIM>>>(d_in, d_out, n);

    cudaSafeCall(hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost));

    hipFree(d_in);
    hipFree(d_out);
}

__global__ void reverse_kernel_inplace(int *in, int n) {
	const int index = threadIdx.x + blockIdx.x * blockDim.x;
	if (index < n/2) {
		int tmp = in[index];
		in[index] = in[n - 1 - index];
		in[n - 1 - index] = tmp;
	}
}

/* In-place reversal of in[] into itself. */
void inplace_reverse( int *in, int n )
{
	int *d_in;
	const size_t size = n * sizeof(*in);

	cudaSafeCall( hipMalloc( (void **)&d_in, size) );	
	cudaSafeCall( hipMemcpy(d_in, in, size, hipMemcpyHostToDevice) );

	reverse_kernel_inplace<<<(n + BLKDIM - 1)/BLKDIM, BLKDIM>>>(d_in, n);

	cudaSafeCall(hipMemcpy(in, d_in, size, hipMemcpyDeviceToHost));

	hipFree(d_in);
}

void fill( int *x, int n )
{
    int i;
    for (i=0; i<n; i++) {
        x[i] = i;
    }
}

int check( const int *x, int n )
{
    int i;
    for (i=0; i<n; i++) {
        if (x[i] != n - 1 - i) {
            fprintf(stderr, "Test FAILED: x[%d]=%d, expected %d\n", i, x[i], n-1-i);
            return 0;
        }
    }
    printf("Test OK\n");
    return 1;
}

int main( int argc, char* argv[] )
{
    int *in, *out;
    int n = 1024*1024;
    const int MAX_N = 512*1024*1024;

    if ( argc > 2 ) {
        fprintf(stderr, "Usage: %s [n]\n", argv[0]);
        return EXIT_FAILURE;
    }

    if ( argc > 1 ) {
        n = atoi(argv[1]);
    }

    if ( n > MAX_N ) {
        fprintf(stderr, "FATAL: input too large (maximum allowed length is %d)\n", MAX_N);
        return EXIT_FAILURE;
    }

    const size_t SIZE = n * sizeof(*in);

    /* Allocate in[] and out[] */
    in = (int*)malloc(SIZE);
    assert(in != NULL);
    out = (int*)malloc(SIZE);
    assert(out != NULL);
    fill(in, n);

    printf("Reverse %d elements... ", n);
    reverse(in, out, n);
    check(out, n);

    printf("In-place reverse %d elements... ", n);
    inplace_reverse(in, n);
    check(in, n);

    /* Cleanup */
    free(in);
    free(out);

    return EXIT_SUCCESS;
}
